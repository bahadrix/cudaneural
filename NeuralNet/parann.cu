#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <time.h>
#include <stdio.h>
#include "parann.cuh";

//Sigmoid function
__device__ float sigmoid(float x) {
	return 1.0 / (1.0 + exp(-x));
}

//Derivative of sigmoid function
__device__ float d_sigmoid(float x) {
	return x * (1 - x);
}

__global__ void train(
	float *input, float *hidden, float *output, // node values
	unsigned int inputSize, unsigned int hiddenSize, unsigned int outputSize,  // node counts
	float *weights_i2h, float *weights_h2o,  // weight infos
	float *trainingInput, float *trainingOutput,
	unsigned int epochCount) {

}



__global__ void f1(float *input, float *hidden, float *output, // node values
	unsigned int inputSize, unsigned int hiddenSize, unsigned int outputSize,  // node counts
	float *weights_i2h, float *weights_h2o,  // weight infos
	unsigned char *trainingInput, unsigned char *trainingOutput,
	unsigned int epochCount, unsigned int iteration) {

	int id = blockDim.x  * blockIdx.x + threadIdx.x;
	

	int from = id / hiddenSize;
	int to = id % hiddenSize;

	
	
	
	float value = trainingInput[iteration * inputSize + from] * weights_i2h[inputSize * to + from];

	atomicAdd(&hidden[to], value);

	//input[from] = trainingInput
}

void setupNN2(NN2* nn2);
void randomizeWeights(NN2* nn2);
int trainWithGPU(NN2* nn2, unsigned char *trainingInput, unsigned char *trainingOutput, int epoch);
bool cudaCheck(hipError_t, char*);



int main() {


	cout << "SETUP PHASE" << LINE;	
	cout << "Setting up training set for " << TRAIN_SIZE << " elements....";
	srand(time(NULL));

	//create  training dataset
	unsigned char inputArray[TRAIN_SIZE * (INPUT_COUNT + 1)];
	unsigned char outputArray[TRAIN_SIZE * (OUTPUT_COUNT)];

	

	for(int t = 0; t < TRAIN_SIZE; t++) {
		
		inputArray[INPUT_COUNT*t + 0] = rand() % 2;			
		inputArray[INPUT_COUNT*t + 1] = rand() % 2;			
		inputArray[INPUT_COUNT*t + 2] = 1; //bias
		
		outputArray[t] = inputArray[INPUT_COUNT*t + 0] TEST_OPERATOR inputArray[INPUT_COUNT*t + 1];
		
	}
	cout << "OK\n" << "Setting up neural network [" << INPUT_COUNT << "i, " << HIDDEN_COUNT << "h]....";
	
	//Setup neural network
	NN2 nn2;
	setupNN2(&nn2);

	//Iterasyon dizileri
	unsigned char inputSet[INPUT_COUNT + 1];
	unsigned char outputSet[OUTPUT_COUNT];
	
	cout << "OK\n";
	

	//initialize the GPU
	//hipError_t initGPU();
	bool errorExist = 0;
	cout << "Initializing device..";
	errorExist |= cudaCheck(hipSetDevice(0),"");

	

	trainWithGPU(&nn2,inputArray, outputArray, MAX_EPOCH);


	errorExist |= cudaCheck(hipDeviceReset(),"Device reset");


	clock_t start = clock();
	cout << LINE << "EXECUTION PHASE" << LINE;
	cout << "Training started [" << MAX_EPOCH << " epoch]...";
	if(!errorExist) {
		cout << "\n Completed with error";
	}
	getchar();
	
	return 0;
}



int trainWithGPU(NN2* nn2, unsigned char *trainingInput, unsigned char *trainingOutput, int epoch) {
	unsigned char *d_trainingInput, *d_trainingOutput;
	float *d_inputArray, *d_hiddenArray, *d_outputArray;
	float *d_weight_i2h, *d_weight_h2o;
	bool errStat = 1;

	cout << "OK\n" << "Allocating memory on GPU..";


	// ALLOCATING MEMORY
	errStat &= cudaCheck( // Allocate input
		hipMalloc((void**)&d_inputArray, nn2->inputCount * sizeof(float)),
		"Memory allocate error: input");

	errStat &= cudaCheck( // Allocate hidden
		hipMalloc((void**)&d_hiddenArray, nn2->hiddenCount * sizeof(float)),
		"Memory allocate error: hidden");

	errStat &= cudaCheck( // Allocate output
		hipMalloc((void**)&d_outputArray, nn2->outputCount * sizeof(float)),
		"Memory allocate error: output");

	errStat &= cudaCheck( // Allocate i2h
		hipMalloc((void**)&d_weight_i2h, 2 * nn2->inputCount * nn2->hiddenCount * sizeof(float)),
		"Memory allocate error: i2h weights");

	errStat &= cudaCheck( // Allocate h2o
		hipMalloc((void**)&d_weight_h2o, 2 * nn2->hiddenCount * nn2->outputCount * sizeof(float)),
		"Memory allocate error: h2o weights");

	errStat &= cudaCheck( // Allocate trainingInput
		hipMalloc((void**)&d_trainingInput, TRAIN_SIZE * nn2->inputCount * sizeof(float)),
		"Memory allocate error: trainingInput");

	errStat &= cudaCheck( // Allocate trainingOutput
		hipMalloc((void**)&d_trainingOutput, TRAIN_SIZE * nn2->outputCount * sizeof(float)),
		"Memory allocate error: trainingOutput");

	
	// COPY DATA ------------------------------------------------------------------
	//hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	
	cout << "OK\nTransferring structure..";

	errStat |= cudaCheck( // Copy input
		hipMemcpy(d_inputArray,nn2->input, nn2->inputCount * sizeof(float), hipMemcpyHostToDevice),
		"Memory allocate error: copying input");

	errStat |= cudaCheck( // Copy hidden
		hipMemcpy(d_hiddenArray,nn2->hidden, nn2->hiddenCount * sizeof(float), hipMemcpyHostToDevice),
		"Memory allocate error: copying hidden");

	errStat |= cudaCheck( // Copy output
		hipMemcpy(d_outputArray,nn2->output, nn2->outputCount * sizeof(float), hipMemcpyHostToDevice),
		"Memory allocate error: copying input");

	errStat |= cudaCheck( // Copy i2h weights
		hipMemcpy(d_weight_i2h,nn2->weight_i2h, 2 * nn2->inputCount * nn2->hiddenCount * sizeof(float), hipMemcpyHostToDevice),
		"Memory allocate error: copying i2hw");

	errStat |= cudaCheck( // Copy h2o weights
		hipMemcpy(d_weight_h2o,nn2->weight_h2o, 2 * nn2->hiddenCount * nn2->outputCount * sizeof(float), hipMemcpyHostToDevice),
		"Memory allocate error: copying i2hw");

	cout << "OK\n" << "Transferring training data..";
	
	errStat |= cudaCheck( // Copy output
		hipMemcpy(d_trainingInput,trainingInput, TRAIN_SIZE * nn2->inputCount * sizeof(unsigned char), hipMemcpyHostToDevice),
		"Memory allocate error: copying input");
	
	errStat |= cudaCheck( // Copy output
		hipMemcpy(d_trainingOutput,trainingOutput, TRAIN_SIZE * nn2->outputCount * sizeof(unsigned char), hipMemcpyHostToDevice),
		"Memory allocate error: copying output");
	
	cout << "OK\n";

	//-----


	int totalWork = TRAIN_SIZE * MAX_EPOCH;
	
	int i2hLinkCount = nn2->inputCount * nn2->hiddenCount;
	int h2oLinkCount = nn2->hiddenCount * nn2->outputCount;
	int it = 0;

	f1<<<totalWork/BLOCK_SIZE, BLOCK_SIZE>>>(
		d_inputArray, d_hiddenArray, d_outputArray,
		nn2->inputCount, nn2->hiddenCount, nn2->outputCount,
		d_weight_i2h, d_weight_h2o,
		d_trainingInput, d_trainingOutput,
		MAX_EPOCH, it);

	errStat |= cudaCheck(hipGetLastError(), "Kernel execution error");
	errStat |= cudaCheck(hipDeviceSynchronize(), "Device synchronize error");

	hipFree(d_inputArray);
	hipFree(d_hiddenArray);
	hipFree(d_outputArray);
	hipFree(d_weight_i2h);
	hipFree(d_weight_h2o);
	hipFree(d_trainingInput);
	hipFree(d_trainingOutput);

	return 0;

}



void setupNN2(NN2* nn2) {

	nn2->inputCount = INPUT_COUNT + 1; 
	nn2->hiddenCount = HIDDEN_COUNT + 1;
	nn2->outputCount = OUTPUT_COUNT;

	//Allocate the memory ***
	nn2->input = (float*)malloc(nn2->inputCount * sizeof(float)); // +1 for bias
	nn2->hidden = (float*)malloc(nn2->hiddenCount * sizeof(float));
	nn2->output = (float*)malloc(nn2->outputCount * sizeof(float));

	// 20 * 8 = 160 Byte'� bir arada veremeyecekse ne baslarim oyle bellege
	nn2->weight_i2h = (float*)calloc(2 * nn2->hiddenCount * nn2->inputCount, sizeof(float));
	nn2->weight_h2o = (float*)calloc(2 * nn2->outputCount * nn2->hiddenCount, sizeof(float));
	
	//Set activation function
	//nn2->activator = &activator;
	//nn2->delta = &delta;

	cout << "OK\n" << "Randomizing weights..";
	
	//Initialize the weights	
	randomizeWeights(nn2);

}

void randomizeWeights(NN2* nn2) {
	

	for(int i = 0; i < nn2->inputCount; i++) {
		for(int h = 0; h < nn2->hiddenCount; h++) {
			// for accessing second layer: (nn2->inputCount * nn2->hiddenCount * layernum) + nn2->inputCount * h + i
			nn2->weight_i2h[nn2->inputCount * h + i] = (RANDOM_float * 4) - 2;
		}
	}

	for(int h = 0; h < nn2->hiddenCount; h++) {
		for(int o = 0; o < nn2->outputCount; o++) {
			nn2->weight_h2o[nn2->hiddenCount * o + h] = (RANDOM_float * 4) - 2;
		}
	}

}

bool cudaCheck(hipError_t cudaStatus, char* errorStr) {
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CUDA ERROR: %s", errorStr);
		return false;
    }
	return true;
}