
#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>

using namespace std;
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}


__global__ void parasumKernel(float *input,unsigned int inputSize, float *output) {

	extern __shared__ float shared[];

	//initialize
	float* stepSize = &shared[inputSize + 0];
	float* step = &shared[inputSize + 1];

	if(threadIdx.x == inputSize) {
		*stepSize = 1;
		*step = 1;
	} else {
		shared[threadIdx.x] = input[threadIdx.x];
	}
	__syncthreads();

	*output = 2;
	atomicAdd(&input[0], 1);


}



int main() {
	const int inputSize = 100;
	float output;
	cout << "Generating input numbers from [1 to " << inputSize << "] ";
	
	float input[inputSize];

	for(int i = 0; i < inputSize; i++) {
		input[i] = i + 1;
	}

	float* d_input;
	float* d_output;
	hipError_t cudaStatus;

	cout << "OK\n Initializing GPU..";
	cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
	cout << "OK\n Allocating GPU memory..";

	cudaStatus = hipMalloc((void**)&d_input, inputSize * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&d_output, sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cout << "OK\n Transferring input to GPU..";

	
    cudaStatus = hipMemcpy(d_input, input, inputSize * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	cout << "OK\n Launching kernel on GPU..";
	//Release the kraken!
	// +1 thread for write step size; +1 for step
	parasumKernel<<<1, inputSize + 2, (inputSize * sizeof(float))+2>>>(d_input,inputSize, d_output);

	  // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(&output, d_output, sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cout << "OK\n Result: " << output;



Error:
    hipFree(d_input);
    hipFree(d_output);
    
	getchar();

    return cudaStatus;
}

int oldmain()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
